#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdio>
#include<hip/hip_runtime.h>
using namespace std;


// __device__ : Functie executata pe GPU, apelabila doar de pe GPU
// __host__ : Functie executata pe host, apelabila doar de pe host
// __global__ : Functie executata pe GPU, apelabila doar de pe host


__constant__ float c_gaussian[64];



void computeGaussianKernelCuda(float delta, int radius)
{
	float h_gaussian[64];
	for (int i = 0; i < 2 * radius + 1; ++i)
	{
		float x = i - radius;
		h_gaussian[i] = expf(-(x * x) / (2.0f * delta * delta));
	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_gaussian), h_gaussian, sizeof(float)*(2 * radius + 1));
}

// Functie ce calculeaza distanta euclidiana dintre 2 puncte cu 4 coordonate
__device__  float euclideanLength(float3 a, float3 b, float d)
{
	float mod = (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y) + (b.z - a.z) * (b.z - a.z) ;
	return expf(-mod / (2.0f * d * d));
}


__device__  float3 multiplyCuda(float a, float3 b)
{
	return { a * b.x, a * b.y, a * b.z};
}


__device__  float3 addCuda(float3 a, float3 b)
{
	return { a.x + b.x, a.y + b.y, a.z + b.z};
}

__global__ void bilateralFilterKernel(float3*  deviceInput, float3*  deviceOutput, float euclideanDelta, int width, int height, int filterRadius)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // dimensiunea x a id-ului global 
	int idy = blockIdx.y * blockDim.y + threadIdx.y; // dimensiunea y a id-ului global 

	if ((idx < width) && (idy < height))
	{
		float sum = 0.0f;
		float3 t = { 0.f, 0.f, 0.f };
		int position = idy * width + idx;
		float3 center = deviceInput[position];
		int r = filterRadius;

		float domainDistance = 0.0f, colorDistance = 0.0f, factor = 0.0f;

		for (int i = -r; i <= r; ++i)
		{
			int currentY = idy + i;
			// verificam ca pozitiile y ale pixelilor invecinati sa nu depaseasca marginile matricei
			if (currentY < 0)
				currentY = 0;
			else if (currentY >= height)
				currentY = height - 1;

			for (int j = -r; j <= r; ++j)
			{
				// verificam ca pozitiile x ale pixelilor invecinati sa nu depaseasca marginile matricei
				int currentX = idx + j;
				if (currentX < 0)
					currentX = 0;
				else if (currentX >= width)
					currentX = width - 1;

				float3 currentPixel = deviceInput[currentY * width + currentX];
				domainDistance = c_gaussian[r + i] * c_gaussian[r + j];
				colorDistance = euclideanLength(currentPixel, center, euclideanDelta);
				factor = domainDistance * colorDistance;
				sum += factor;
				t = addCuda(t, multiplyCuda(factor, currentPixel));
			}
		}

		deviceOutput[position] = multiplyCuda(1.f / sum, t);
	}
}



void bilateralFilterCuda(float3*  hostInput, float3*  hostOutput, float euclideanDelta, int width, int height, int filterRadius)
{
	// compute the gaussian kernel for the current radius and delta
	computeGaussianKernelCuda(euclideanDelta, filterRadius);

	int inputBytes = width * height * sizeof(float3);
	int outputBytes = inputBytes;

	float3* deviceInput, *deviceOutput;
	hipMalloc<float3>(&deviceInput, inputBytes);
	hipMalloc<float3>(&deviceOutput, outputBytes);


	hipMemcpy(deviceInput, hostInput, inputBytes, hipMemcpyHostToDevice); // copiem datele in memoria GPU


	dim3 block(8, 8); //definim un bloc de 8x8 threaduri
	dim3 grid((width + block.x - 1) , (height + block.y - 1) );// definim gridul astfel incat sa acopere toata imaginea


	bilateralFilterKernel << <grid, block >> > (deviceInput, deviceOutput, euclideanDelta, width, height, filterRadius);


	hipDeviceSynchronize(); // blocheaza CPU pana toate apelurile CUDA se finalizeaza

	hipMemcpy(hostOutput, deviceOutput, outputBytes, hipMemcpyDeviceToHost); // copiem rezultatul de pe GPU pe CPU

	hipFree(deviceInput);
	hipFree(deviceOutput);
}